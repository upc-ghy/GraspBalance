#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "group_points_gpu.h"


__global__ void group_points_grad_kernel_fast(int b, int c, int n, int npoints, int nsample, 
    const float *__restrict__ grad_out, const int *__restrict__ idx, float *__restrict__ grad_points) {
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_idx = index / nsample;
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int sample_idx = index % nsample;
    grad_out += bs_idx * c * npoints * nsample + c_idx * npoints * nsample + pt_idx * nsample + sample_idx;
    idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx; 
    
    atomicAdd(grad_points + bs_idx * c * n + c_idx * n + idx[0] , grad_out[0]);
}

void group_points_grad_kernel_launcher_fast(int b, int c, int n, int npoints, int nsample, 
    const float *grad_out, const int *idx, float *grad_points) {
    hipError_t err;
    dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_grad_kernel_fast<<<blocks, threads>>>(b, c, n, npoints, nsample, grad_out, idx, grad_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void group_points_kernel_fast(int b, int c, int n, int npoints, int nsample, 
    const float *__restrict__ points, const int *__restrict__ idx, float *__restrict__ out) {
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_idx = index / nsample;
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int sample_idx = index % nsample;

    idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx; 
    int in_idx = bs_idx * c * n + c_idx * n + idx[0];
    int out_idx = bs_idx * c * npoints * nsample + c_idx * npoints * nsample + pt_idx * nsample + sample_idx;

    out[out_idx] = points[in_idx];
}


void group_points_kernel_launcher_fast(int b, int c, int n, int npoints, int nsample, 
    const float *points, const int *idx, float *out) {
    hipError_t err;
    dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_kernel_fast<<<blocks, threads>>>(b, c, n, npoints, nsample, points, idx, out);
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
